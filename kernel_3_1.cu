﻿#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

#include <time.h>
#include <cmath>

#define CUDA_CHECK_RETURN(value){\
		hipError_t _m_cudaStat = value;\
		if (_m_cudaStat != hipSuccess) {\
			fprintf(stderr, "Error %s at line %d in file %s\n",\
			hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);\
			exit(1);\
		} }

typedef unsigned int uint;

//const uint V_SIZE = 1024*1024*512; // 1 << 29
//const uint V_SIZE = 1024*1024*32; // 1 << 25
const uint V_SIZE = 1024 * 1024; // 1 << 20
const uint MAX_NUM_OF_THREADS = 1024;

__global__ void d_vector_add(int* a, int* b) {

    int idX = threadIdx.x + blockDim.x * blockIdx.x;

    if (idX < V_SIZE) {
        a[idX] = idX;
        b[idX] = V_SIZE - idX;
        a[idX] += b[idX];
    }
}

__host__ float d_test(int threadsPerBlock, int numOfBlocks) {

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    int *da, *db;
    float elapsedTime;

    CUDA_CHECK_RETURN(hipMalloc((void**)&da, V_SIZE * sizeof(int)));
    CUDA_CHECK_RETURN(hipMalloc((void**)&db, V_SIZE * sizeof(int)));

    hipEventRecord(start, 0);

    d_vector_add<<<numOfBlocks,
                   threadsPerBlock>>>(da, db);

    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);

    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    CUDA_CHECK_RETURN(hipGetLastError());

    hipEventElapsedTime(&elapsedTime, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(da);
    hipFree(db);

    return elapsedTime;
}


int main() {

    int device;
    hipDeviceProp_t prop;

    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);

    float time;

    int numOfBlocks,
        result,        // Occupancy in terms of active blocks
        activeWarps,
        maxWarps;

    for (uint threadsPerBlock = 32; threadsPerBlock <= MAX_NUM_OF_THREADS; threadsPerBlock += 32) {
       numOfBlocks = V_SIZE / threadsPerBlock;
       time = d_test(threadsPerBlock, numOfBlocks);
       printf("threads_per_block = %4d; num_of_blocks = %8d; time = %4.2f ms; ", threadsPerBlock, numOfBlocks, time);

       // Reference: https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#occupancy-calculator
       // Reference: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__HIGHLEVEL.html#group__CUDART__HIGHLEVEL_1g5a5d67a3c907371559ba692195e8a38c
       // 
       // Theoretical Occupancy
       // The following code sample calculates the occupancy of MyKernel.
       // It then reports the occupancy level with the ratio between concurrent warps versus maximum warps per multiprocessor.
       hipOccupancyMaxActiveBlocksPerMultiprocessor(
           &result,
           (void*)d_vector_add,
           threadsPerBlock,
           0);
       activeWarps = result * threadsPerBlock / prop.warpSize;
       maxWarps = prop.maxThreadsPerMultiProcessor / prop.warpSize;

       printf("\tActive warps - %d; Max warps - %d; ", activeWarps, maxWarps);
       printf("Theoretical Occupancy : %.2lf%\n", (double)activeWarps / maxWarps * 100);

    }

    printf("\n");
    return 0;
}