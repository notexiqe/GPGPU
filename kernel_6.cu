#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define N (1024 * 1024)
#define FULL_DATA_SIZE (N * 20)

__global__ void kernel(int* a, int* b, int* c) {

    int i = threadIdx.x;

    c[i] = a[i] + b[i];
}

__global__ void kernel2(int* a, int* b, int* c) {
    __shared__ float cache[256];
    int i, tid = threadIdx.x + blockIdx.x * blockDim.x, cacheIndex = threadIdx.x, temp = 0;

    while (tid < N) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
    cache[cacheIndex] = temp;
    __syncthreads();

    i = blockDim.x / 2;

    while (i != 0) {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }
    if (cacheIndex == 0)
        c[blockIdx.x] = cache[0];
}


int main() {
    srand(time(NULL));

    hipDeviceProp_t prop;
    int i, res, whichDevice;
    int* dev_a, * dev_a_p, * h_a, * h_b, * h_a_p, * h_b_p, * h_c_p;
    int* dev_a0, * dev_b0, * dev_c0, * dev_a1, * dev_b1, * dev_c1;
    float elapsed_time;
    hipStream_t stream0, stream1;
    hipEvent_t start, stop;

    hipGetDevice(&whichDevice);
    hipGetDeviceProperties(&prop, whichDevice);
    if (!prop.deviceOverlap) {
        printf("Device does not support overlapping\n");
        return 0;
    }
    hipEventCreate(&start);
    hipEventCreate(&stop);

    h_a = (int*)malloc(FULL_DATA_SIZE * sizeof(int));
    h_b = (int*)malloc(FULL_DATA_SIZE * sizeof(int));
    for (int i = 0; i < FULL_DATA_SIZE; i++)
        h_a[i] = rand() % 10000;
    hipMalloc((void**)&dev_a, FULL_DATA_SIZE * sizeof(int));
    hipEventRecord(start, 0);
    hipMemcpy(dev_a, h_a, FULL_DATA_SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("Time host->device: %f\n", elapsed_time);

    hipEventRecord(start, 0);
    hipMemcpy(h_b, dev_a, FULL_DATA_SIZE * sizeof(int), hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("Time device->host: %f\n", elapsed_time);

    hipHostAlloc((void**)&h_a_p, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&h_b_p, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&h_c_p, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
    hipMalloc((void**)&dev_a_p, FULL_DATA_SIZE * sizeof(int));
    for (i = 0; i < FULL_DATA_SIZE; i++)
        h_a_p[i] = rand() % 10000;
    hipEventRecord(start, 0);
    hipMemcpy(dev_a_p, h_a_p, FULL_DATA_SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("Time paged-locked host->device: %f\n", elapsed_time);

    hipEventRecord(start, 0);
    hipMemcpy(h_b_p, dev_a_p, FULL_DATA_SIZE * sizeof(int), hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("Time paged-locked device->host: %f\n", elapsed_time);
    hipFree(dev_a);
    hipFree(dev_a_p);


    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);
    hipMalloc((void**)&dev_a0, FULL_DATA_SIZE * sizeof(int));
    hipMalloc((void**)&dev_b0, FULL_DATA_SIZE * sizeof(int));
    hipMalloc((void**)&dev_c0, FULL_DATA_SIZE * sizeof(int));
    hipMalloc((void**)&dev_a1, FULL_DATA_SIZE * sizeof(int));
    hipMalloc((void**)&dev_b1, FULL_DATA_SIZE * sizeof(int));
    hipMalloc((void**)&dev_c1, FULL_DATA_SIZE * sizeof(int));
    for (i = 0; i < FULL_DATA_SIZE; i++) {
        h_a_p[i] = i;
        h_b_p[i] = i;
    }
    hipEventRecord(start, 0);
    for (i = 0; i < FULL_DATA_SIZE; i += N * 2) {
        hipMemcpyAsync(dev_a0, h_a_p + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(dev_a1, h_a_p + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);
        hipMemcpyAsync(dev_b0, h_b_p + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(dev_b1, h_b_p + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);
        kernel << < 1, N, 0, stream0 >> > (dev_a0, dev_b0, dev_c0);
        kernel << < 1, N, 0, stream1 >> > (dev_a1, dev_b1, dev_c1);
        hipMemcpyAsync(h_c_p + i, dev_c0, N * sizeof(int), hipMemcpyDeviceToHost, stream0);
        hipMemcpyAsync(h_c_p + i + N, dev_c1, N * sizeof(int), hipMemcpyDeviceToHost, stream1);
    }
    hipStreamSynchronize(stream0);
    hipStreamSynchronize(stream1);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("Time addition vectors: %f\n", elapsed_time);

    hipEventRecord(start, 0);
    for (i = 0; i < FULL_DATA_SIZE; i += N * 2) {
        hipMemcpyAsync(dev_a0, h_a_p + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(dev_a1, h_a_p + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);
        hipMemcpyAsync(dev_b0, h_b_p + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(dev_b1, h_b_p + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);
        kernel2 << < 16, N, 0, stream0 >> > (dev_a0, dev_b0, dev_c0);
        kernel2 << < 16, N, 0, stream1 >> > (dev_a1, dev_b1, dev_c1);
        hipMemcpyAsync(h_c_p + i, dev_c0, N * sizeof(int), hipMemcpyDeviceToHost, stream0);
        hipMemcpyAsync(h_c_p + i + N, dev_c1, N * sizeof(int), hipMemcpyDeviceToHost, stream1);
    }
    hipStreamSynchronize(stream0);
    hipStreamSynchronize(stream1);
    for (i = 0, res = 0; i < FULL_DATA_SIZE; i++)
        res += h_c_p[i];
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("Time multiplication vectors: %f\n", elapsed_time);

    hipFree(dev_a0);
    hipFree(dev_a1);
    hipFree(dev_b0);
    hipFree(dev_b1);
    hipFree(dev_c0);
    hipFree(dev_c1);
    hipStreamDestroy(stream0);
    hipStreamDestroy(stream1);
    hipFree(h_a_p);
    hipFree(h_b_p);
    hipFree(h_c_p);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("\n");
    return 0;
}