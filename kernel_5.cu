#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define M_PI 3.14159265358979323846
#define COEF 48
#define VERTCOUNT COEF * COEF * 2
#define RADIUS 10.0f
#define FGSIZE 20
#define FGSHIFT FGSIZE / 2
#define IMIN(A,B) (A<B?A:B)
#define THREADSPERBLOCK 256
#define BLOCKSPERGRID IMIN(32, (VERTCOUNT+THREADSPERBLOCK-1)/THREADSPERBLOCK)

typedef float(*ptr_f)(float, float, float);

struct Vertex {
    float x, y, z;
};

__constant__ Vertex vert[VERTCOUNT];

texture<float, 3, hipReadModeElementType> df_tex;

hipArray* df_Array = 0;

float func(float x, float y, float z) {
    return (0.5 * sqrtf(15.0 / M_PI)) * (0.5 * sqrtf(15.0 / M_PI)) *
        z * z * y * y * sqrtf(1.0f - z * z / RADIUS / RADIUS) / RADIUS / RADIUS / RADIUS / RADIUS;
}

__device__ float func1(float x, float y, float z) {

    return (0.5f * sqrtf(15.0f / M_PI)) * (0.5f * sqrtf(15.0f / M_PI)) * z * z * y * y * sqrtf(1.0f - z * z / RADIUS / RADIUS) / RADIUS / RADIUS / RADIUS / RADIUS; 
}

float check(Vertex* v, ptr_f f) {

    float sum = 0.0f;

    for (int i = 0; i < VERTCOUNT; ++i)
        sum += f(v[i].x, v[i].y, v[i].z);
    return sum;
}

void calc_f(float* arr_f, int x_size, int y_size, int z_size, ptr_f f) {

    for (int x = 0; x < x_size; ++x) {
        for (int y = 0; y < y_size; ++y) {
            for (int z = 0; z < z_size; ++z) {
                arr_f[z_size * (x * y_size + y) + z] = f(x - FGSHIFT, y - FGSHIFT, z - FGSHIFT);
            }
        }
    }
}

void init_vertices() {

    Vertex* temp_vert = (Vertex*)malloc(sizeof(Vertex) * VERTCOUNT);
    int i = 0;

    for (int iphi = 0; iphi < 2 * COEF; ++iphi) {
        for (int ipsi = 0; ipsi < COEF; ++ipsi, ++i) {
            float phi = iphi * M_PI / COEF;
            float psi = ipsi * M_PI / COEF;
            temp_vert[i].x = RADIUS * sinf(psi) * cosf(phi);
            temp_vert[i].y = RADIUS * sinf(psi) * sinf(phi);
            temp_vert[i].z = RADIUS * cosf(psi);
        }
    }
    printf("sumCheck = %f\n", check(temp_vert, &func) * M_PI * M_PI / COEF / COEF);
    hipMemcpyToSymbol(HIP_SYMBOL(vert), temp_vert, sizeof(Vertex) * VERTCOUNT, 0, hipMemcpyHostToDevice);
    free(temp_vert);
    return;
}

void init_texture(float* df_h) {

    const hipExtent volumeSize = make_hipExtent(FGSIZE, FGSIZE, FGSIZE);

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipMalloc3DArray(&df_Array, &channelDesc, volumeSize);
    hipMemcpy3DParms cpyParams = { 0 };
    cpyParams.srcPtr = make_hipPitchedPtr((void*)df_h, volumeSize.width * sizeof(float), volumeSize.width, volumeSize.height);
    cpyParams.dstArray = df_Array;
    cpyParams.extent = volumeSize;
    cpyParams.kind = hipMemcpyHostToDevice;
    hipMemcpy3D(&cpyParams);
    df_tex.normalized = false;
    df_tex.filterMode = hipFilterModeLinear;
    df_tex.addressMode[0] = hipAddressModeClamp;
    df_tex.addressMode[1] = hipAddressModeClamp;
    df_tex.addressMode[2] = hipAddressModeClamp;
    hipBindTextureToArray(df_tex, df_Array, channelDesc);
    return;
}

void release_texture() {

    hipUnbindTexture(df_tex);
    hipFreeArray(df_Array);
    return;
}

__device__ float interpol(float x, float y, float z, float* arr) {

    float x_r[2], y_r[2], z_r[2], denominator, res = 0.0f;;

    x_r[0] = x - 1;
    x_r[1] = x + 1;
    y_r[0] = y - 1;
    y_r[1] = y + 1;
    z_r[0] = z - 1;
    z_r[1] = z + 1;
    denominator = (x_r[1] - x_r[0]) * (y_r[1] - y_r[0]) * (z_r[1] - z_r[0]);
    res += (arr[FGSIZE * ((int)x_r[0] * FGSIZE + (int)y_r[0]) + (int)z_r[0]] * (x_r[1] - x) * (y_r[1] - y) * (z_r[1] - z)) / denominator;
    res += (arr[FGSIZE * ((int)x_r[0] * FGSIZE + (int)y_r[0]) + (int)z_r[1]] * (x_r[1] - x) * (y_r[1] - y) * (z - z_r[0])) / denominator;
    res += (arr[FGSIZE * ((int)x_r[0] * FGSIZE + (int)y_r[1]) + (int)z_r[0]] * (x_r[1] - x) * (y - y_r[0]) * (z_r[1] - z)) / denominator;
    res += (arr[FGSIZE * ((int)x_r[0] * FGSIZE + (int)y_r[1]) + (int)z_r[1]] * (x_r[1] - x) * (y - y_r[0]) * (z - z_r[0])) / denominator;
    res += (arr[FGSIZE * ((int)x_r[1] * FGSIZE + (int)y_r[0]) + (int)z_r[0]] * (x - x_r[0]) * (y_r[1] - y) * (z_r[1] - z)) / denominator;
    res += (arr[FGSIZE * ((int)x_r[1] * FGSIZE + (int)y_r[0]) + (int)z_r[1]] * (x - x_r[0]) * (y_r[1] - y) * (z - z_r[0])) / denominator;
    res += (arr[FGSIZE * ((int)x_r[1] * FGSIZE + (int)y_r[1]) + (int)z_r[0]] * (x - x_r[0]) * (y - y_r[0]) * (z_r[1] - z)) / denominator;
    res += (arr[FGSIZE * ((int)x_r[1] * FGSIZE + (int)y_r[1]) + (int)z_r[1]] * (x - x_r[0]) * (y - y_r[0]) * (z - z_r[0])) / denominator;
    //printf("%f\n", res);
    return res;
}

__global__ void kernel(float* a) {

    __shared__ float cache[THREADSPERBLOCK];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    float x = vert[tid].x + FGSHIFT + 0.5f;
    float y = vert[tid].y + FGSHIFT + 0.5f;
    float z = vert[tid].z + FGSHIFT + 0.5f;

    cache[cacheIndex] = tex3D(df_tex, z, y, x);
    __syncthreads();
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (cacheIndex < s)
            cache[cacheIndex] += cache[cacheIndex + s];
        __syncthreads();
    }
    if (cacheIndex == 0)
        a[blockIdx.x] = cache[0];
    return;
}

__global__ void kernel2(float* a, float* val) {

    __shared__ float cache[THREADSPERBLOCK];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    float x = vert[tid].x + FGSHIFT + 0.5f;
    float y = vert[tid].y + FGSHIFT + 0.5f;
    float z = vert[tid].z + FGSHIFT + 0.5f;

    cache[cacheIndex] = interpol(x, y, z, val);
    __syncthreads();
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (cacheIndex < s)
            cache[cacheIndex] += cache[cacheIndex + s];
        __syncthreads();
    }
    if (cacheIndex == 0)
        a[blockIdx.x] = cache[0];
}

int main() {

    float* arr = (float*)malloc(sizeof(float) * FGSIZE * FGSIZE * FGSIZE);
    float* values, * sum_dev, * sum = (float*)malloc(sizeof(float) * BLOCKSPERGRID), s;
    float sum1, sum2;
    hipMalloc((void**)&sum_dev, sizeof(float) * BLOCKSPERGRID);
    init_vertices();
    calc_f(arr, FGSIZE, FGSIZE, FGSIZE, &func);
    hipMalloc((void**)&values, sizeof(float) * FGSIZE * FGSIZE * FGSIZE);
    hipMemcpy(values, arr, sizeof(float) * FGSIZE * FGSIZE * FGSIZE, hipMemcpyHostToDevice);
    init_texture(arr);
    kernel << <BLOCKSPERGRID, THREADSPERBLOCK >> > (sum_dev);
    hipDeviceSynchronize();
    hipMemcpy(sum, sum_dev, sizeof(float) * BLOCKSPERGRID, hipMemcpyDeviceToHost);
    s = 0.0f;
    for (int i = 0; i < BLOCKSPERGRID; i++)
        s += sum[i];
    sum1 = s * M_PI * M_PI / COEF / COEF;
    printf("sum1 = %f\n", sum1);
    kernel2 << <BLOCKSPERGRID, THREADSPERBLOCK >> > (sum_dev, values);
    hipDeviceSynchronize();
    hipMemcpy(sum, sum_dev, sizeof(float) * BLOCKSPERGRID, hipMemcpyDeviceToHost);
    s = 0.0f;
    for (int i = 0; i < BLOCKSPERGRID; i++)
        s += sum[i];
    sum2 = s * M_PI * M_PI / COEF / COEF;
    printf("sum2 = %f\n", sum2);
    if (sum1 > sum2)
        printf("Sum2 faster sum1 by %f\n", (sum1 - sum2));
    else
        printf("Sum1 faster sum2 by %f\n", (sum2 - sum1));
    hipFree(sum_dev);
    free(sum);
    release_texture();
    free(arr);
    return 0;
}